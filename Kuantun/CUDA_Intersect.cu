#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CUDA_Intersect.cuh"
#include <stdio.h>

#define INFINITE 1e30

__device__ vertex operator +(vertex &v0, vertex &v1) {
	vertex ret = { v0.x + v1.x, v0.y + v1.y, v0.z + v1.z };
	return ret;
}

__device__ vertex operator - (vertex &v0, vertex &v1) {
	vertex ret = { v0.x - v1.x, v0.y - v1.y, v0.z - v1.z };
	return ret;
}

__device__ float dot(vertex &v0, vertex &v1) {
	return v0.x*v1.x + v0.y * v1.y + v0.z * v1.z;
}

__device__ vertex operator *(vertex &v0, float &f) {
	vertex ret = { v0.x * f, v0.y * f, v0.z * f };
	return ret;
}

__device__ vertex cross(vertex &v0, vertex &v1) {
	vertex ret = { v0.y * v1.z - v0.z * v1.y , v0.z * v1.x - v0.x * v1.z, v0.x  * v1.y - v0.y * v1.x };
	return ret;
}

__global__ void moller_trumbore(vertex raydir, vertex rayorig, float* vertexBuffer, float* hit, float* distance, float* uv, unsigned int triangles){
	unsigned int triangleNumber = blockIdx.x * blockDim.x + threadIdx.x;
	if (triangleNumber < triangles) {
		unsigned int toff = 9 * triangleNumber;
		distance[triangleNumber] = -1.0f;
		vertex v0 = { vertexBuffer[toff] , vertexBuffer[toff + 1], vertexBuffer[toff + 2] };
		vertex e1 = { vertexBuffer[toff + 3] , vertexBuffer[toff + 4], vertexBuffer[toff + 5] };
		vertex e2 = { vertexBuffer[toff + 6] , vertexBuffer[toff + 7], vertexBuffer[toff + 8] };
		vertex pvec = cross(raydir, e2);
		float inv_det = 1.0f / dot(e1, pvec);
		vertex tvec = rayorig - v0;
		float ut = dot(tvec, pvec) * inv_det;
		if (ut >= 0 && ut <= 1) {
			vertex qvec = cross(tvec, e1);
			float vt = dot(raydir, qvec) * inv_det;
			if (vt >= 0 && ut + vt <= 1) {
				float dist = dot(e2, qvec) * inv_det;
				vertex inter = raydir * dist + rayorig;
				unsigned int hitoff = 3 * triangleNumber;
				hit[hitoff] = inter.x;
				hit[hitoff + 1] = inter.y;
				hit[hitoff + 2] = inter.z;
				distance[triangleNumber] = dist;
				uv[2 * triangleNumber] = ut;
				uv[2 * triangleNumber + 1] = vt;
			}
		}
	}
}

Vector3D* CUDA_Intersect(Vector3D &rayorig, Vector3D &raydir, unsigned int &index, float &u, float &v, float* vertexBuffer, unsigned int vertexCount){

	float *hits_out_dev, *distance_out_dev, *uv_out_dev;

	size_t sizeh, sized, sizeuv;
	sizeh = (sizeof(float) * vertexCount) / 3;
	sized = (sizeof(float) * vertexCount) / 9;
	sizeuv = (2 * sizeof(float) * vertexCount) / 9;

	hipError_t ce;
	do {
		ce = hipMalloc(&hits_out_dev, sizeh);
	} while (ce != hipSuccess);
	do {
		ce = hipMalloc(&distance_out_dev, sized);
	} while (ce != hipSuccess);
	do {
		ce = hipMalloc(&uv_out_dev, sizeuv);
	} while (ce != hipSuccess);

	dim3 tamGrid((vertexCount / (CUDABLOCKSIZE * 9)), 1);
	dim3 tamBlock(CUDABLOCKSIZE, 1);

	vertex rd, ro;
	rd = { raydir.x,raydir.y,raydir.z };
	ro = { rayorig.x,rayorig.y,rayorig.z };

	moller_trumbore <<< tamGrid, tamBlock >>> (rd, ro, vertexBuffer, hits_out_dev, distance_out_dev, uv_out_dev, vertexCount/9);

	hipDeviceSynchronize();

	float * hits, *distance, *uv;
	hits =  (float*) malloc(sizeh);
	distance = (float*)malloc(sized);
	uv = (float*)malloc(sizeuv);

	hipMemcpy(hits, hits_out_dev,sizeh, hipMemcpyDeviceToHost);
	hipMemcpy(distance, distance_out_dev,sized, hipMemcpyDeviceToHost);
	hipMemcpy(uv, uv_out_dev,sizeuv, hipMemcpyDeviceToHost);

	float dist = INFINITE;
	Vector3D* hit = new Vector3D();
	index = 0;
	for (unsigned int i = 0; i < vertexCount / 9; i++) {
		if (distance[i] > 1e-7 && distance[i] < dist) {
			hit->x = hits[3*i];
			hit->y = hits[3*i + 1];
			hit->z = hits[3 * i + 2];
			u = uv[2 * i];
			v = uv[2 * i + 1];
			dist = distance[i];
			index = i;
		}
	}

	if (dist == -1.0f) {
		delete hit;
		hit = NULL;
	}

	delete[] hits;
	delete[] distance;
	delete[] uv;

	hipFree(hits_out_dev);
	hipFree(distance_out_dev);
	hipFree(uv_out_dev);

	return hit;
}

